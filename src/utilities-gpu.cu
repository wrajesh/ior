#include "hip/hip_runtime.h"
/*
  This file contains CUDA code for creating and checking memory patterns on the device.
*/
#include <hip/hip_runtime.h>

#ifdef HAVE_CONFIG_H
#include "config.h"
#endif

#include <stdint.h>

#include "iordef.h"

#define RANDALGO_GOLDEN_RATIO_PRIME        0x9e37fffffffc0001UL

__global__ 
void cu_generate_memory_timestamp(uint64_t * buf, size_t length, int rand_seed, uint64_t pretendRank){
  size_t pos = blockIdx.x * blockDim.x + threadIdx.x;
  if(pos < length){
    buf[pos] = pretendRank | rand_seed + pos;
  }
}

__global__ 
void cu_generate_memory_incompressible(uint64_t * buf, size_t length, uint64_t seed){
  size_t pos = blockIdx.x * blockDim.x + threadIdx.x;
  if(pos < length){
    buf[pos] = seed | pos;
  }
}

__global__ 
void cu_verify_memory_timestamp(uint64_t item, uint64_t * buf, size_t length, int rand_seed, uint64_t pretendRank, int * errors){
  size_t pos = blockIdx.x * blockDim.x + threadIdx.x;
  if(pos < length){
    int correct = buf[pos] == (pretendRank | rand_seed + pos);
    if(! correct){
      *errors = 1; // it isn't thread safe but one error reported is enough
    }
  }
}

extern "C" void generate_memory_pattern_gpu(char * buf, size_t bytes, int rand_seed, int pretendRank, ior_dataPacketType_e dataPacketType){    
  size_t blocks = (bytes+2047)/2048;
  size_t threads = 256;
  switch(dataPacketType){
    case(DATA_RANDOM):
      // Nothing to do, will work on updates
      break;
    case(DATA_INCOMPRESSIBLE):{      
      cu_generate_memory_incompressible<<<blocks, threads>>>((uint64_t*) buf, bytes/sizeof(uint64_t), rand_seed + pretendRank);
      break;
    }case(DATA_OFFSET):{
    }case(DATA_TIMESTAMP):{
      cu_generate_memory_timestamp<<<blocks, threads>>>((uint64_t*) buf, bytes/sizeof(uint64_t), rand_seed, ((uint64_t) pretendRank) << 32);
      break;
    }
  }
}

extern "C" void update_write_memory_pattern_gpu(uint64_t item, char * buf, size_t bytes, int rand_seed, int rank, ior_dataPacketType_e dataPacketType){
  // nothing to do for dataPacketType == DATA_TIMESTAMP, i.e., won't be called for this parameter
  size_t blocks = (bytes+2047)/2048;
  size_t threads = 256;
}

extern "C" int verify_memory_pattern_gpu(uint64_t item, char * buffer, size_t bytes, int rand_seed, int pretendRank, ior_dataPacketType_e dataPacketType){
  int errors = 0;
  size_t blocks = (bytes+2047)/2048;
  size_t threads = 256;  
  int * derror_found;
  hipMalloc(&derror_found, sizeof(int));
  hipMemcpy(derror_found, & errors, sizeof(int), hipMemcpyHostToDevice);
  if(dataPacketType == DATA_TIMESTAMP){
    cu_verify_memory_timestamp<<<blocks, threads>>>(item, (uint64_t*) buffer, bytes/sizeof(uint64_t), rand_seed, ((uint64_t) pretendRank) << 32, derror_found);
  }else if(dataPacketType == DATA_INCOMPRESSIBLE){
    
  }
  hipMemcpy(& errors, derror_found, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(derror_found);
  return errors;
}
